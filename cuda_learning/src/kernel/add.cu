#include "hip/hip_runtime.h"
#include "utils/common.h"
#include <iostream>
#include <stdio.h>

__global__
void helloWorld() {
    printf("Hello world: %d %d %d\n", threadIdx.x, blockIdx.x, blockDim.x);
}

__global__
void cuda_sum_1d(float* A, float* B, float* C, size_t size) {
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__
void cuda_sum_2d(float* A, float*B, float* C, size_t nx, size_t ny) {
    auto idx = blockDim.x * blockIdx.x + threadIdx.x;
    auto idy = blockDim.y * blockIdx.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        auto pos = idx * ny + idy;
        C[pos] = A[pos] + B[pos];
    }
}

void sum1d() {
    auto nElem = 4096 * 4096;
    auto nBytes = nElem * sizeof(float);

    auto A = (float*) malloc(nBytes);
    auto B = (float*) malloc(nBytes);
    auto C = (float*) malloc(nBytes);
    auto C_GPU = (float*) malloc(nBytes);
    initialData(A, nElem);
    initialData(B, nElem);

    auto cpu_start = cpuSecond();
    for (auto i = 0; i < nElem; i++) {
        C[i] = A[i] + B[i];
    }
    std::cout << "CPU cost: " << cpuSecond() - cpu_start << "\n";
    
    float* dA;
    float* dB;
    float* dC;
    CHECK(hipMalloc((float**) &dA, nBytes));
    CHECK(hipMalloc((float**) &dB, nBytes));
    CHECK(hipMalloc((float**) &dC, nBytes));
    CHECK(hipMemcpy(dA, A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, nBytes, hipMemcpyHostToDevice));

    dim3 block(1024);
    std::cout << "block: " << block.x << "," << block.y << "\n";
    dim3 grid = (nElem + block.x - 1) / block.x;
    std::cout << "grid: " << grid.x << "," << grid.y << "\n";

    auto gpu_start = cpuSecond();
    cuda_sum_1d<<<grid, block>>>(dA, dB, dC, nElem);
    CHECK(hipDeviceSynchronize());
    std::cout << "GPU cost: " << cpuSecond() - gpu_start << "\n";

    CHECK(hipMemcpy(C_GPU, dC, nBytes, hipMemcpyDeviceToHost));

    for (auto i = 0; i < nElem; i++) {
        if (C[i] != C_GPU[i]) {
            std::cout << "diff! " << i << "\n";
            break;
        }
    }

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    free(A);
    free(B);
    free(C);
    free(C_GPU);
}

void sum2d() {
    auto nx = (1 << 16);
    auto ny = (1 << 12);
    auto nBytes = nx * ny * sizeof(float);
    auto A = (float*) malloc(nBytes);
    auto B = (float*) malloc(nBytes);
    auto C = (float*) malloc(nBytes);
    auto C_GPU = (float*) malloc(nBytes);
    initialData(A, nx * ny);
    initialData(B, nx * ny);

    float* dA;
    float* dB;
    float* dC;
    CHECK(hipMalloc((float**) &dA, nBytes));
    CHECK(hipMalloc((float**) &dB, nBytes));
    CHECK(hipMalloc((float**) &dC, nBytes));
    CHECK(hipMemcpy(dA, A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, nBytes, hipMemcpyHostToDevice));

    dim3 block(32, 32);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    auto gpu_start = cpuSecond();
    cuda_sum_2d<<<grid, block>>>(dA, dB, dC, nx, ny);
    CHECK(hipDeviceSynchronize());
    std::cout << "GPU cost: " << cpuSecond() - gpu_start << "\n";

    auto cpu_start = cpuSecond();
    for (auto i = 0; i < nx; i++) {
        for (auto j = 0; j < ny; j++) {
            C[i * ny + j] = A[i * ny + j] + B[i * ny + j];
        }
    }
    std::cout << "CPU cost: " << cpuSecond() - cpu_start << "\n";

    CHECK(hipMemcpy(C_GPU, dC, nBytes, hipMemcpyDeviceToHost));

    for (auto i = 0; i < nx; i++) {
        for (auto j = 0; j < ny; j++) {
            if (C[i * ny + j] != C_GPU[i * ny + j]) {
                std::cout << "diff! " << i << "\n";
                break;
            }
        }
    }
    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    free(A);
    free(B);
    free(C);
    free(C_GPU);
}